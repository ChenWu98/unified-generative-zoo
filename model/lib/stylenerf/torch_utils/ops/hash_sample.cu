// Copyright (c) Facebook, Inc. and its affiliates.All Rights Reserved


// Please refer to original code: https://github.com/NVlabs/instant-ngp
// and the pytorch wrapper from https://github.com/ashawkey/torch-ngp

#include <stdint.h>
#include <hip/hip_runtime.h>


#include <algorithm>
#include <stdexcept>

#include <cstdio>

template <typename T>
__host__ __device__ T div_round_up(T val, T divisor) {
	return (val + divisor - 1) / divisor;
}


template <uint32_t D>
__device__ uint32_t fast_hash(const uint32_t pos_grid[D]) {
	static_assert(D <= 7, "fast_hash can only hash up to 7 dimensions.");

	// While 1 is technically not a good prime for hashing (or a prime at all), it helps memory coherence
	// and is sufficient for our use case of obtaining a uniformly colliding index from high-dimensional
	// coordinates.
	constexpr uint32_t primes[7] = { 1, 19349663, 83492791, 25165843, 6291469, 12582917, 3145739 };

	uint32_t result = 0;
	#pragma unroll
	for (uint32_t i = 0; i < D; ++i) {
		result ^= pos_grid[i] * primes[i];
	}

	return result;
}


template <uint32_t D, uint32_t C>
__device__ uint32_t get_grid_index(const uint32_t ch, const uint32_t hashmap_size, const uint32_t resolution, const uint32_t pos_grid[D], const uint32_t mode) {
	uint32_t stride = 1;
	uint32_t index = 0;
	
    switch(mode) {
        case 0:   // fast-hash
            #pragma unroll
            for (uint32_t d = 0; d < D && stride <= hashmap_size; d++) {
                // printf("get_grid_index d=%d, pos_grid[d]=%d, stride=%d, reso=%d\n", d, pos_grid[d], stride, resolution);
                index += pos_grid[d] * stride;
                stride *= (resolution + 1);
            }
            if (stride > hashmap_size) {
                //printf("hash because %d > %d\n", stride, hashmap_size);
                index = fast_hash<D>(pos_grid);
                //printf("hashed (%d, %d) = %d to %d in %d\n", pos_grid[0], pos_grid[1], pos_grid[0] + resolution * pos_grid[1], index % hashmap_size, hashmap_size);
            }
            index = index % hashmap_size; break;
        
        case 1:   // grid-hash
            uint32_t h_res = (uint32_t)cbrtf(hashmap_size);
            #pragma unroll
            for (uint32_t d = 0; d < D; d++) {
                index += (pos_grid[d] % h_res) * stride;
                stride *= h_res;
            }
            break;
    }
	return index * C + ch;
}


template <uint32_t D, uint32_t C>
__global__ void kernel_grid(
    const float * __restrict__ inputs, 
    const float * __restrict__ grid, 
    const int * __restrict__ offsets, 
    float * outputs, 
    const float beta,
    uint32_t B, uint32_t N, 
    uint32_t L, uint32_t H,
    const bool calc_grad_inputs, 
    float * dy_dx,
    uint32_t mode) {
    
    const uint32_t b = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (b >= N) return;

    const uint32_t level    = blockIdx.y;
    const uint32_t batch_id = blockIdx.z;
    const uint32_t batch_offset_grid   = offsets[L] * batch_id;
    const uint32_t batch_offset_inputs = N * batch_id;

    // locate
    grid    += ((uint32_t)offsets[level] + batch_offset_grid) * C;
    inputs  += ( b + batch_offset_inputs) * D;
    outputs += ((b + batch_offset_inputs) * L + level) * C;

    const uint32_t hashmap_size = offsets[level + 1] - offsets[level];
    
    // const float scale = exp2f(level) * H - 1.0f;
    const float scale = powf(beta, level) * H - 1.0f;
    const uint32_t resolution = (uint32_t)ceil(scale) + 1;
    // const float scale = powf(beta, level) * H;
    // const uint32_t resolution = (uint32_t)ceil(scale);

    // calculate coordinate
    float pos[D];
    uint32_t pos_grid[D];

    #pragma unroll
    for (uint32_t d = 0; d < D; d++) {
        pos[d] = inputs[d] * scale + 0.5f;
        pos_grid[d] = floorf(pos[d]);
        pos[d] -= (float)pos_grid[d];
    }

    // printf("[b=%d, l=%d] pos=(%f, %f)+(%d, %d) scale=%f \n", b, level, pos[0], pos[1], pos_grid[0], pos_grid[1], scale);

    // interpolate
    #pragma unroll
    for (uint32_t idx = 0; idx < (1 << D); idx++) {
        float w = 1;
        uint32_t pos_grid_local[D];

        #pragma unroll
        for (uint32_t d = 0; d < D; d++) {
            if ((idx & (1 << d)) == 0) {
                w *= 1 - pos[d];
                pos_grid_local[d] = pos_grid[d];
            } else {
                w *= pos[d];
                pos_grid_local[d] = pos_grid[d] + 1;
            }
        }

        uint32_t index = get_grid_index<D, C>(0, hashmap_size, resolution, pos_grid_local, mode);

        #pragma unroll
        for (uint32_t ch = 0; ch < C; ch++) {
            outputs[ch] += w * grid[index + ch];
        }

        //printf("[b=%d, l=%d] int %d, idx %d, w %f, val %f\n", b, level, idx, index, w, grid[index]);
    }    

    // prepare dy_dx for calc_grad_inputs
    if (calc_grad_inputs) {

        // dy_dx += b * D * L * C + level * D * C; // B N L D C
        dy_dx += ((b + batch_offset_inputs) * L + level) * D * C;

        #pragma unroll
        for (uint32_t gd = 0; gd < D; gd++) {

            #pragma unroll
            for (uint32_t idx = 0; idx < (1 << (D - 1)); idx++) {
                float w = scale;
                uint32_t pos_grid_local[D];

                #pragma unroll
                for (uint32_t nd = 0; nd < D - 1; nd++) {
                    const uint32_t d = nd > gd ? nd + 1 : nd;

                    if ((idx & (1 << nd)) == 0) {
                        w *= 1 - pos[d];
                        pos_grid_local[d] = pos_grid[d];
                    } else {
                        w *= pos[d];
                        pos_grid_local[d] = pos_grid[d] + 1;
                    }
                }

                pos_grid_local[gd] = pos_grid[gd];
                uint32_t index_left = get_grid_index<D, C>(0, hashmap_size, resolution, pos_grid_local, mode);
                pos_grid_local[gd] = pos_grid[gd] + 1;
                uint32_t index_right = get_grid_index<D, C>(0, hashmap_size, resolution, pos_grid_local, mode);

                #pragma unroll
                for (uint32_t ch = 0; ch < C; ch++) {
                    dy_dx[gd * C + ch] += w * (grid[index_right + ch] - grid[index_left + ch]);
                }
            }
        }
    }
}


template <uint32_t D, uint32_t C, uint32_t N_C>
__global__ void kernel_grid_backward(
    const float * __restrict__ grad,
    const float * __restrict__ inputs, 
    const float * __restrict__ grid, 
    const int * __restrict__ offsets, 
    float * grad_grid, 
    const float beta,
    uint32_t B, uint32_t N, 
    uint32_t L, uint32_t H,
    uint32_t mode
) {
    const uint32_t b = (blockIdx.x * blockDim.x + threadIdx.x) * N_C / C;
	if (b >= N) return;

    const uint32_t level = blockIdx.y;
    const uint32_t ch = (blockIdx.x * blockDim.x + threadIdx.x) * N_C - b * C;
    const uint32_t batch_id = blockIdx.z;
    const uint32_t batch_offset_grid   = offsets[L] * batch_id;
    const uint32_t batch_offset_inputs = N * batch_id;

    // locate
    grad_grid += ((uint32_t)offsets[level] + batch_offset_grid) * C;
    inputs    += ( b + batch_offset_inputs) * D;
    grad      += ((b + batch_offset_inputs) * L + level) * C + ch;

    const uint32_t hashmap_size = offsets[level + 1] - offsets[level];
    // const float scale = exp2f(level) * H - 1.0f;
    const float scale = powf(beta, level) * H - 1.0f;
    const uint32_t resolution = (uint32_t)ceil(scale) + 1;

    // calculate coordinate
    float pos[D];
    uint32_t pos_grid[D];

    #pragma unroll
    for (uint32_t d = 0; d < D; d++) {
        pos[d] = inputs[d] * scale + 0.5f;
        pos_grid[d] = floorf(pos[d]);
        pos[d] -= (float)pos_grid[d];
    }

    // interpolate
    #pragma unroll
    for (uint32_t idx = 0; idx < (1 << D); idx++) {
        float w = 1;
        uint32_t pos_grid_local[D];

        #pragma unroll
        for (uint32_t d = 0; d < D; d++) {
            if ((idx & (1 << d)) == 0) {
                w *= 1 - pos[d];
                pos_grid_local[d] = pos_grid[d];
            } else {
                w *= pos[d];
                pos_grid_local[d] = pos_grid[d] + 1;
            }
        }
        
        uint32_t index = get_grid_index<D, C>(ch, hashmap_size, resolution, pos_grid_local, mode);

        #pragma unroll
        for (uint32_t c = 0; c < N_C; c++) {
            atomicAdd(&grad_grid[index + c], w * grad[c]);
        }
    }    
}


template <uint32_t D, uint32_t C>
__global__ void kernel_input_backward(
    const float * __restrict__ grad,
    const float * __restrict__ dy_dx,  
    float * grad_inputs,
    uint32_t B, uint32_t N, uint32_t L
) {
    const uint32_t t = threadIdx.x + blockIdx.x * blockDim.x;
    if (t >= N * D) return;

    const uint32_t b = t / D;
    const uint32_t d = t - b * D;
    const uint32_t batch_id = blockIdx.y;
    const uint32_t batch_offset_inputs = N * batch_id;

    grad  += (b + batch_offset_inputs) * L * C;
    dy_dx += (b + batch_offset_inputs) * L * D * C;
    grad_inputs += N * D * batch_id;

    # pragma unroll
    for (int l = 0; l < L; l++) {
        # pragma unroll
        for (int ch = 0; ch < C; ch++) {
            grad_inputs[t] += grad[l * C + ch] * dy_dx[l * D * C + d * C + ch];
        }
    }
}


template <uint32_t D>
void kernel_grid_wrapper(const float *inputs, const float *embeddings, const int *offsets, float *outputs, const float beta, const uint32_t B, const uint32_t N, const uint32_t C, const uint32_t L, const uint32_t H, const bool calc_grad_inputs, float *dy_dx, const uint32_t mode) {
    static constexpr uint32_t N_THREAD = 512;
	const dim3 blocks_hashgrid = { div_round_up(N, N_THREAD), L, B};
    switch (C) {
        case 1: kernel_grid<D, 1><<<blocks_hashgrid, N_THREAD>>>(inputs, embeddings, offsets, outputs, beta, B, N, L, H, calc_grad_inputs, dy_dx, mode); break;
        case 2: kernel_grid<D, 2><<<blocks_hashgrid, N_THREAD>>>(inputs, embeddings, offsets, outputs, beta, B, N, L, H, calc_grad_inputs, dy_dx, mode); break;
        case 4: kernel_grid<D, 4><<<blocks_hashgrid, N_THREAD>>>(inputs, embeddings, offsets, outputs, beta, B, N, L, H, calc_grad_inputs, dy_dx, mode); break;
        case 8: kernel_grid<D, 8><<<blocks_hashgrid, N_THREAD>>>(inputs, embeddings, offsets, outputs, beta, B, N, L, H, calc_grad_inputs, dy_dx, mode); break;
        case 32: kernel_grid<D, 32><<<blocks_hashgrid, N_THREAD>>>(inputs, embeddings, offsets, outputs, beta, B, N, L, H, calc_grad_inputs, dy_dx, mode); break;
        default: throw std::runtime_error{"GridEncoding: C must be 1, 2, 4, 8, 32"};
    }
}

// inputs: [B, D], float, in [0, 1]
// embeddings: [sO, C], float
// offsets: [L + 1], uint32_t
// outputs: [B, L * C], float
// H: base resolution
void hash_encode_forward_cuda(const float *inputs, const float *embeddings, const int *offsets, float *outputs, const float beta, const uint32_t B, const uint32_t N, const uint32_t D, const uint32_t C, const uint32_t L, const uint32_t H, const bool calc_grad_inputs, float *dy_dx, const uint32_t mode) {
    switch (D) {
        case 2: kernel_grid_wrapper<2>(inputs, embeddings, offsets, outputs, beta, B, N, C, L, H, calc_grad_inputs, dy_dx, mode); break;
        case 3: kernel_grid_wrapper<3>(inputs, embeddings, offsets, outputs, beta, B, N, C, L, H, calc_grad_inputs, dy_dx, mode); break;
        default: throw std::runtime_error{"We only support 2D or 3D data for now."};
    }
    
}

template <uint32_t D>
void kernel_grid_backward_wrapper(const float *grad, const float *inputs, const float *embeddings, const int *offsets, float *grad_embeddings, const float beta, const uint32_t B,  const uint32_t N, const uint32_t C, const uint32_t L, const uint32_t H, const bool calc_grad_inputs, float *dy_dx, float *grad_inputs, const uint32_t mode) {
    static constexpr uint32_t N_THREAD = 256;
	const uint32_t N_C = std::min(2u, C); // n_features_per_thread
	const dim3 blocks_hashgrid = {div_round_up(N * C / N_C, N_THREAD), L, B};   // batch x sample x level
    const dim3 input_blocks_hashgrid = {div_round_up(N * D, N_THREAD), B, 1};
    switch (C) {
        case 1: 
            kernel_grid_backward<D, 1, 1><<<blocks_hashgrid, N_THREAD>>>(grad, inputs, embeddings, offsets, grad_embeddings, beta, B, N, L, H, mode); 
            if (calc_grad_inputs) kernel_input_backward<D, 1><<<input_blocks_hashgrid, N_THREAD>>>(grad, dy_dx, grad_inputs, B, N, L);
            break;
        case 2: 
            kernel_grid_backward<D, 2, 2><<<blocks_hashgrid, N_THREAD>>>(grad, inputs, embeddings, offsets, grad_embeddings, beta, B, N, L, H, mode);
            if (calc_grad_inputs) kernel_input_backward<D, 2><<<input_blocks_hashgrid, N_THREAD>>>(grad, dy_dx, grad_inputs, B, N, L);
            break;
        case 4: 
            kernel_grid_backward<D, 4, 2><<<blocks_hashgrid, N_THREAD>>>(grad, inputs, embeddings, offsets, grad_embeddings, beta, B, N, L, H, mode);
            if (calc_grad_inputs) kernel_input_backward<D, 4><<<input_blocks_hashgrid, N_THREAD>>>(grad, dy_dx, grad_inputs, B, N, L);
            break;
        case 8: 
            kernel_grid_backward<D, 8, 2><<<blocks_hashgrid, N_THREAD>>>(grad, inputs, embeddings, offsets, grad_embeddings, beta, B, N, L, H, mode);
            if (calc_grad_inputs) kernel_input_backward<D, 8><<<input_blocks_hashgrid, N_THREAD>>>(grad, dy_dx, grad_inputs, B, N, L);
            break;
        case 32:
            kernel_grid_backward<D, 32, 4><<<blocks_hashgrid, N_THREAD>>>(grad, inputs, embeddings, offsets, grad_embeddings, beta, B, N, L, H, mode);
            if (calc_grad_inputs) kernel_input_backward<D, 32><<<input_blocks_hashgrid, N_THREAD>>>(grad, dy_dx, grad_inputs, B, N, L);
            break;
        default: throw std::runtime_error{"GridEncoding: C must be 1, 2, 4, or 8."};
    }
}


// grad: [B, L * C], float
// inputs: [B, D], float, in [0, 1]
// embeddings: [sO, C], float
// offsets: [L + 1], uint32_t
// grad_embeddings: [sO, C]
// H: base resolution
void hash_encode_backward_cuda(const float *grad, const float *inputs, const float *embeddings, const int *offsets, float *grad_embeddings, const float beta, const uint32_t B,  const uint32_t N, const uint32_t D, const uint32_t C, const uint32_t L, const uint32_t H, const bool calc_grad_inputs, float *dy_dx, float *grad_inputs, const uint32_t mode) {
    switch (D) {
        case 2: kernel_grid_backward_wrapper<2>(grad, inputs, embeddings, offsets, grad_embeddings, beta, B, N, C, L, H, calc_grad_inputs, dy_dx, grad_inputs, mode); break;
        case 3: kernel_grid_backward_wrapper<3>(grad, inputs, embeddings, offsets, grad_embeddings, beta, B, N, C, L, H, calc_grad_inputs, dy_dx, grad_inputs, mode); break;
        default: throw std::runtime_error{"We only support 2D or 3D data for now."};
    }
}