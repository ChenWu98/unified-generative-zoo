#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates.All Rights Reserved


#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <torch/torch.h>
#include <torch/extension.h>

#include "utils.h"


template <typename T>
__host__ __device__ T div_round_up(T val, T divisor) {
	return (val + divisor - 1) / divisor;
}


template <uint32_t S>
__global__ void kernel_topp_masking(
    const int * __restrict__ sorted_indices,
    const float * __restrict__ sorted_weights, 
    bool *output_mask, 
    const float p, const uint32_t B, 
    const uint32_t N, const uint32_t D) {

    const uint32_t b = blockIdx.x * blockDim.x + threadIdx.x;
    if (b >= N) return;

    const uint32_t batch_id = blockIdx.y;
    
    // locate
    sorted_weights += (b + batch_id * N) * D;
    sorted_indices += (b + batch_id * N) * D;
    output_mask += (b + batch_id * N) * D;
    
    float w_sum = 0;

    #pragma unroll
    for (uint32_t d = 0; d < S; d++){
        if (d >= D) break;
        w_sum += sorted_weights[d];
        output_mask[sorted_indices[d]] = true;
        if (w_sum >= p) break;
    }
    }

void topp_masking_cuda(
    const int *sorted_indices, 
    const float *sorted_weights, bool *output_mask, 
    const float p, const uint32_t B, const uint32_t N, const uint32_t D) {
    static constexpr uint32_t N_THREAD = 512;
    const dim3 blocks = {div_round_up(N, N_THREAD), B, 1};
    if (D < 8)        kernel_topp_masking<8><<<  blocks, N_THREAD>>>(sorted_indices, sorted_weights, output_mask, p, B, N, D);
    else if (D < 16)  kernel_topp_masking<16><<< blocks, N_THREAD>>>(sorted_indices, sorted_weights, output_mask, p, B, N, D);
    else if (D < 32)  kernel_topp_masking<32><<< blocks, N_THREAD>>>(sorted_indices, sorted_weights, output_mask, p, B, N, D);
    else if (D < 64)  kernel_topp_masking<64><<< blocks, N_THREAD>>>(sorted_indices, sorted_weights, output_mask, p, B, N, D);
    else if (D < 128) kernel_topp_masking<128><<<blocks, N_THREAD>>>(sorted_indices, sorted_weights, output_mask, p, B, N, D);
    else if (D < 256) kernel_topp_masking<256><<<blocks, N_THREAD>>>(sorted_indices, sorted_weights, output_mask, p, B, N, D);
    else throw std::runtime_error{"# of sampled points should not exceed 256"};

}

void topp_masking(
    at::Tensor sorted_indices, at::Tensor sorted_weights, at::Tensor output_mask, 
    const float p, const uint32_t B, const uint32_t N, const uint32_t D) {
    CHECK_CUDA(sorted_indices);
    CHECK_CUDA(sorted_weights);
    CHECK_CUDA(output_mask);
  
    CHECK_CONTIGUOUS(sorted_indices);
    CHECK_CONTIGUOUS(sorted_weights);
    CHECK_CONTIGUOUS(output_mask);

    CHECK_IS_FLOAT(sorted_weights);
    CHECK_IS_INT(sorted_indices);
    
    topp_masking_cuda(sorted_indices.data_ptr<int>(), sorted_weights.data_ptr<float>(), output_mask.data_ptr<bool>(), p, B, N, D);
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("topp_masking", &topp_masking, "topp masking");
}
